#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>

#define MAX_VERTICES 1024
#define MAX_EDGES 5000

// Kernel to calculate triangle support for each edge
__global__ void calculateSupport(int *adj, int *edges, int *sup, int numEdges, int numVertices) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEdges) {
        int u = edges[idx * 2];
        int v = edges[idx * 2 + 1];
        int count = 0;
        for (int i = 0; i < numVertices; i++) {
            if (adj[u * numVertices + i] && adj[v * numVertices + i]) {
                count++; // Count common neighbors (triangles)
            }
        }
        sup[idx] = count;
    }
}

// Host function for Naive k-Truss
void naiveKTruss(int *adj, int *edges, int numVertices, int numEdges, int k, int *edgeDel, int *numTrusses, int *remainingEdges, int *totalTriangles) {
    int *d_adj, *d_edges, *d_sup;
    hipMalloc(&d_adj, numVertices * numVertices * sizeof(int));
    hipMalloc(&d_edges, numEdges * 2 * sizeof(int));
    hipMalloc(&d_sup, numEdges * sizeof(int));

    hipMemcpy(d_adj, adj, numVertices * numVertices * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges, edges, numEdges * 2 * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (numEdges + blockSize - 1) / blockSize;
    bool changed = true;

    while (changed) {
        changed = false;
        hipMemset(d_sup, 0, numEdges * sizeof(int));
        calculateSupport<<<numBlocks, blockSize>>>(d_adj, d_edges, d_sup, numEdges, numVertices);
        hipDeviceSynchronize();

        int *sup = (int *)malloc(numEdges * sizeof(int));
        hipMemcpy(sup, d_sup, numEdges * sizeof(int), hipMemcpyDeviceToHost);

        for (int e = 0; e < numEdges; e++) {
            if (edgeDel[e] == -1 && sup[e] < k - 2) {
                edgeDel[e] = k - 1;
                changed = true;
            }
        }
        free(sup);
    }

    // Post-processing to count trusses, remaining edges, and triangles
    *remainingEdges = 0;
    int *ktrussAdj = (int *)malloc(numVertices * numVertices * sizeof(int));
    memset(ktrussAdj, 0, numVertices * numVertices * sizeof(int));
    for (int e = 0; e < numEdges; e++) {
        if (edgeDel[e] == -1) {
            (*remainingEdges)++;
            int u = edges[e * 2];
            int v = edges[e * 2 + 1];
            ktrussAdj[u * numVertices + v] = 1;
            ktrussAdj[v * numVertices + u] = 1;
        }
    }
    *totalTriangles = 0;
    for (int u = 0; u < numVertices; u++) {
        for (int v = u + 1; v < numVertices; v++) {
            if (ktrussAdj[u * numVertices + v]) {
                for (int w = v + 1; w < numVertices; w++) {
                    if (ktrussAdj[v * numVertices + w] && ktrussAdj[u * numVertices + w]) {
                        (*totalTriangles)++;
                    }
                }
            }
        }
    }
    *numTrusses = (*remainingEdges > 0) ? 1 : 0;

    free(ktrussAdj);
    hipFree(d_adj);
    hipFree(d_edges);
    hipFree(d_sup);
}

int main() {
    srand(time(NULL));
    int numVertices = 100;
    int numEdges = 0;
    int adj[MAX_VERTICES * MAX_VERTICES];
    int edges[MAX_EDGES * 2];
    int edgeDel[MAX_EDGES];

    // Initialize adjacency matrix with 70% density
    for (int i = 0; i < numVertices * numVertices; i++) {
        int row = i / numVertices;
        int col = i % numVertices;
        adj[i] = (rand() % 100 < 70) ? 1 : 0; // 70% edge probability
        if (row == col) adj[i] = 0; // No self-loops
        adj[col * numVertices + row] = adj[i]; // Symmetry
    }

    // Generate edge list
    for (int i = 0; i < numVertices; i++) {
        for (int j = i + 1; j < numVertices; j++) {
            if (adj[i * numVertices + j]) {
                if (numEdges < MAX_EDGES) {
                    edges[numEdges * 2] = i;
                    edges[numEdges * 2 + 1] = j;
                    edgeDel[numEdges] = -1;
                    numEdges++;
                }
            }
        }
    }

    int k = 4;
    int numTrusses = 0, remainingEdges = 0, totalTriangles = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    naiveKTruss(adj, edges, numVertices, numEdges, k, edgeDel, &numTrusses, &remainingEdges, &totalTriangles);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Naive k-Truss Time: %f ms\n", milliseconds);
    printf("Number of %d-Trusses: %d\n", k, numTrusses);
    printf("Remaining Edges in %d-Truss: %d\n", k, remainingEdges);
    printf("Total Triangles in %d-Truss: %d\n", k, totalTriangles);
    printf("\nEdge Deletion Status (first 10 edges as sample):\n");
    for (int i = 0; i < (numEdges > 10 ? 10 : numEdges); i++) {
        printf("Edge (%d, %d): %s\n", edges[i * 2], edges[i * 2 + 1], edgeDel[i] == -1 ? "Kept" : "Deleted");
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
